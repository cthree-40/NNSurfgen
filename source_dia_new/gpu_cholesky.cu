//nvcc -gencode=arch=compute_35,code=sm_35 -lcublas -ccbin "/work1/soft/intel/composer_xe_2013.0.079/bin/intel64/icc" -c this.cu
#include<stdio.h>
#include<stdlib.h>
#include<assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
extern "C" void cuda_cholesky_(int *n, double *a, double *b, int *info)
{
    int lwork;
    int n1,n2;
    double *d_work = NULL;
    double *d_A = NULL;
    double *d_B = NULL;
    int *devInfo = NULL;
    hipsolverHandle_t cusolverH = NULL;

    n1=*n;
    n2=n1;

    hipsolverDnCreate(&cusolverH);
    hipMalloc((void**)&devInfo, sizeof(int));
    hipMalloc((void**)&d_A , sizeof(double)*n1*n2);
    hipMalloc((void**)&d_B , sizeof(double)*n1);
    hipMemcpy(d_A, a, sizeof(double)*n1*n2, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeof(double)*n1, hipMemcpyHostToDevice);

    //query working space
    hipsolverDnDpotrf_bufferSize(cusolverH,HIPBLAS_FILL_MODE_UPPER,n1,d_A,n2,&lwork);
    hipMalloc((void**)&d_work, sizeof(double)*lwork);

    hipsolverDnDpotrf(cusolverH,HIPBLAS_FILL_MODE_UPPER,n1,d_A,n2,d_work,lwork,devInfo);
    hipMemcpy(info,devInfo,sizeof(int),hipMemcpyDeviceToHost);

    if(*info == 0){
        hipsolverDnDpotrs(cusolverH,HIPBLAS_FILL_MODE_UPPER,n1,1,d_A,n2,d_B,n1,devInfo);
        hipMemcpy(info,devInfo,sizeof(int),hipMemcpyDeviceToHost);
        if(*info == 0) hipMemcpy(b,d_B,n1*sizeof(double),hipMemcpyDeviceToHost);
    }

    if(d_A) hipFree(d_A);
    if(d_B) hipFree(d_B);
    if(d_work) hipFree(d_work);
    if(devInfo) hipFree(devInfo);
    if(cusolverH) hipsolverDnDestroy(cusolverH);
}
