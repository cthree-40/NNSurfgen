//nvcc -gencode=arch=compute_35,code=sm_35 -lcublas -ccbin "/work1/soft/intel/composer_xe_2013.0.079/bin/intel64/icc" -c this.cu
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
// u = 'u', t = 't', A(nt,nw), C(nw,nw), C=A'*A
extern "C" void gpu_dsyrk_(char *u, char *n, int *nw, int *nt, double *alpha, double *a, int *lda, double *beta, double *c, int *ldc)
{
  int N1,N2;
  double *dA,*dC;
  hipblasHandle_t handle;
  N1=*nt;
  N2=*nw;
  hipMalloc((void **)&dA,N1*N2*sizeof(double));
  hipMalloc((void **)&dC,N2*N2*sizeof(double));
  hipblasCreate(&handle);
  hipMemcpy(dA,a,N1*N2*sizeof(double),hipMemcpyHostToDevice);
  hipblasDsyrk(handle,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N,N2,N1,alpha,dA,N2,beta,dC,N2);
  hipMemcpy(c,dC,N2*N2*sizeof(double),hipMemcpyDeviceToHost);
  hipFree(dA);
  hipFree(dC);
  hipblasDestroy(handle);
}
